
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <vector>

#define CSC(call)       \
do {                    \
    hipError_t status = call;          \
    if  (status != hipSuccess) {       \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));   \
        exit(0);                        \
    }                                   \
} while (0)

#define BENCHMARK

#ifdef BENCHMARK
hipEvent_t benchmarkStart, benchmarkStop;

void startBenchmark() {
    CSC(hipEventCreate(&benchmarkStart));
    CSC(hipEventCreate(&benchmarkStop));
    CSC(hipEventRecord(benchmarkStart));
}

void stopBenchmark() {
    CSC(hipEventRecord(benchmarkStop));
    CSC(hipEventSynchronize(benchmarkStop));
    float time;
    CSC(hipEventElapsedTime(&time, benchmarkStart, benchmarkStop));
    CSC(hipEventDestroy(benchmarkStart));
    CSC(hipEventDestroy(benchmarkStop));
    std::cout << "time = " << time << " ms\n";
}
#endif /* BENCHMARK */

__global__ void kernel(hipTextureObject_t tex, uchar4 *out, int width, int height) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;
    int x, y;
    uchar4 p;
    for(y = idy; y < height; y += offsety)
        for(x = idx; x < width; x += offsetx) {
            double w[2][2];

            // color convert
            for (int i = 0; i < 2; ++i) {
                for (int j = 0; j < 2; ++j) {
                    p = tex2D<uchar4>(tex, x+i, y+j);
                    w[i][j] = 0.299*p.x + 0.587*p.y + 0.114*p.z;
                }
            }

            // roberts method
            double gx = w[1][1] - w[0][0];
            double gy = w[1][0] - w[0][1];
            int gf = min(255, int(sqrt(gx*gx + gy*gy)));

            out[y*width + x] = make_uchar4(gf, gf, gf, gf);
        }
}

void readData(std::string& filename, int& w, int& h, uchar4** data) {
    FILE *f = fopen(filename.c_str(), "rb");
    fread(&w, sizeof(int), 1, f);
    fread(&h, sizeof(int), 1, f);
    *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
    fread(*data, sizeof(uchar4), w * h, f);
    fclose(f);
}

void writeData(std::string& filename, int w, int h, uchar4* data) {
    FILE *f = fopen(filename.c_str(), "wb");
    fwrite(&w, sizeof(int), 1, f);
    fwrite(&h, sizeof(int), 1, f);
    fwrite(data, sizeof(uchar4), w * h, f);
    fclose(f);
}

const int X_BLOCKS = 16;
const int X_THREADS = 16;
const int Y_BLOCKS = 32;
const int Y_THREADS = 32;

int main() {
    int w, h;
    uchar4 *pixels = nullptr;
    std::string input_filename, output_filename;
    std::cin >> input_filename >> output_filename;
    readData(input_filename, w, h, &pixels);

    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpy2DToArray(arr, 0, 0, pixels, w * sizeof(uchar4), w * sizeof(uchar4), h, hipMemcpyHostToDevice));

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arr;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = false;

    hipTextureObject_t tex = 0;
    CSC(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

    uchar4 *dev_out;
    CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));

#ifdef BENCHMARK
    startBenchmark();
#endif /* BENCHMARK */

    kernel<<< dim3(X_BLOCKS, X_THREADS), dim3(Y_BLOCKS, Y_THREADS) >>>(tex, dev_out, w, h);

    CSC(hipDeviceSynchronize());
    CSC(hipGetLastError());

#ifdef BENCHMARK
    stopBenchmark();
#endif /* BENCHMARK */

    CSC(hipMemcpy(pixels, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

    CSC(hipDestroyTextureObject(tex));
    CSC(hipFreeArray(arr));
    CSC(hipFree(dev_out));

#ifndef BENCHMARK
    writeData(output_filename, w, h, pixels);
#endif /* BENCHMARK */

    free(pixels);
    return 0;
}
