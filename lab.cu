
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CSC(call)       \
do {                    \
    hipError_t status = call;          \
    if  (status != hipSuccess) {       \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));   \
        exit(0);                        \
    }                                   \
} while (0)

// #define BENCHMARK


__global__ void kernel(double *v1, double *v2, double *res, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    while(idx < n) {
        res[idx] = v1[idx] - v2[idx];
        idx += offset;
    }
}

double *readVector(int n) {
    double *v = (double *)malloc(sizeof(double) * n);
    for (int i = 0; i < n; ++i) {
        scanf("%lf", &v[i]);
    }
    return v;
}

void printVector(double *v, int n) {
    for (int i = 0; i < n; ++i) {
        printf("%.10lf ", v[i]);
    }
    printf("\n");
}

const int BLOCKS = 1024;
const int THREADS = 1024;

int main() {
    int n;
    scanf("%d", &n);
    double *v1 = readVector(n);
    double *v2 = readVector(n);
    double *v_res = (double *)malloc(sizeof(double) * n);

    double *dev_v1, *dev_v2, *dev_res;
    CSC(hipMalloc(&dev_v1, sizeof(double) * n));
    CSC(hipMalloc(&dev_v2, sizeof(double) * n));
    CSC(hipMalloc(&dev_res, sizeof(double) * n));
    CSC(hipMemcpy(dev_v1, v1, sizeof(double) * n, hipMemcpyHostToDevice));
    CSC(hipMemcpy(dev_v2, v2, sizeof(double) * n, hipMemcpyHostToDevice));

#ifdef BENCHMARK
    cudaEvent_t start, stop;
    CSC(cudaEventCreate(&start));
    CSC(cudaEventCreate(&stop));
    CSC(cudaEventRecord(start));
#endif /* BENCHMARK */

    kernel<<<BLOCKS, THREADS>>>(dev_v1, dev_v2, dev_res, n);

    CSC(hipDeviceSynchronize());
    CSC(hipGetLastError());

#ifdef BENCHMARK
    CSC(cudaEventRecord(stop));
    CSC(cudaEventSynchronize(stop));
    float time;
    CSC(cudaEventElapsedTime(&time, start, stop));
    CSC(cudaEventDestroy(start));
    CSC(cudaEventDestroy(stop));
    printf("time = %f ms\n", time);
#endif /* BENCHMARK */

    CSC(hipMemcpy(v_res, dev_res, sizeof(double) * n, hipMemcpyDeviceToHost));

#ifndef BENCHMARK
    printVector(v_res, n);
#endif

    free(v1);
    free(v2);
    free(v_res);
    CSC(hipFree(dev_v1));
    CSC(hipFree(dev_v2));
    CSC(hipFree(dev_res));
    return 0;
}
